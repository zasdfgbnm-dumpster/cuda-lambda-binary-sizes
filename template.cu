
#include <hip/hip_runtime.h>
#define LAMBDA_TYPE __device__ __host__

template<long long i, typename func_t>
__global__ void elementwise_kernel(float *a, float *b, float *c, int N, func_t f) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = f(a[idx], b[idx]);
    }
}

template<long long l = 0, long long r = 128>
void _add(float *a, float *b, float *c, float alpha, int N) {
    if constexpr (l == r) {
        elementwise_kernel<l><<<(N + 255) / 256, 256>>>(a, b, c, N,
            [=] LAMBDA_TYPE (float a, float b) { return a + alpha * b; }
        );
    } else if constexpr (l + 1 == r) {
        elementwise_kernel<l><<<(N + 255) / 256, 256>>>(a, b, c, N,
            [=] LAMBDA_TYPE (float a, float b) { return a + b; }
        );
        elementwise_kernel<r><<<(N + 255) / 256, 256>>>(a, b, c, N,
            [=] LAMBDA_TYPE (float a, float b) { return a + b; }
        );
    } else {
        constexpr long long m = (l + r) / 2;
        _add<l, m>(a, b, c, alpha, N);
        _add<m, r>(a, b, c, alpha, N);
    }
}

void add(float *a, float *b, float *c, int N) {
    _add(a, b, c, 1.23, N);
}
